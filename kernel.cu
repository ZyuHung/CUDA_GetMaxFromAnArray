#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>

__global__ void GetMaxNum_kernel(int *num, int *max)
{
	__shared__ int *best;
	best = (int*)malloc(sizeof(int)* 20);
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int tid_idx = 2;
	//************ Method 2 (Better) ************
	while ((tid_idx<= pow((float)2,(int)sqrt((float)20)+1)) &&(tid*tid_idx<20))
	{
		if (num[tid*tid_idx] < num[tid*tid_idx + tid_idx / 2])
		{
			num[tid*tid_idx] = num[tid*tid_idx + tid_idx / 2];
			best[tid*tid_idx] = tid*tid_idx + tid_idx / 2;
		}
		tid_idx *= 2;
		__syncthreads();
	}

	//************ Method 1 ************
	/*for (int i = 1; i <= 8; i *= 2)
	{
		if (tid < (20 / tid_idx))
		{
			num[tid*tid_idx] = (num[tid*tid_idx] >= num[tid*tid_idx + i]) ? num[tid*tid_idx] : num[tid*tid_idx + i];
		}
		tid_idx *= 2;
		__syncthreads();
	}
	tid_idx /= 2;
	if (20 != tid_idx)
	{
		if (tid == tid_idx)
		{
			num[0] = (num[0] >= num[tid]) ? num[0] : num[tid];
		}
	}*/

	*max = num[0];
	printf("best num idx is :%d\n", best[0]);
}

int main()
{
	int *a = new int[20];
	int *num, *b, *max=new int(0);
	for (int  i = 0; i < 20; i++)
	{
		a[i] = i*i;
	}
	hipMalloc(&num, sizeof(int)* 20);
	hipMalloc(&b, sizeof(int));
	hipMemcpy(num, a, sizeof(int)* 20, hipMemcpyHostToDevice);

	GetMaxNum_kernel << <1, 20 >> >(num, b);

	hipDeviceSynchronize();

	hipMemcpy(max, b, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(&num);
	hipFree(&b);

	printf("the MAX num is: %d\n", *max);

	getchar();
	return 0;
}