#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>

__global__ void GetMaxNum_kernel(int *num, int *max)
{
	//__shared__ int MAX;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int tid_idx = 2;
	for (int i = 1; i <= 8; i *= 2)
	{
		if (tid < (20 / tid_idx))
		{
			num[tid*tid_idx] = (num[tid*tid_idx] >= num[tid*tid_idx + i]) ? num[tid*tid_idx] : num[tid*tid_idx + i];
		}
		tid_idx *= 2;
		__syncthreads();
	}
	tid_idx /= 2;
	if (20 != tid_idx)
	{
		if (tid == tid_idx)
		{
			num[0] = (num[0] >= num[tid]) ? num[0] : num[tid];
		}
	}
	*max = num[0];
}

int main()
{
	int *a = new int[20];
	int *num, *b, *max=new int(0);
	for (int  i = 0; i < 20; i++)
	{
		a[i] = i*i;
	}
	hipMalloc(&num, sizeof(int)* 20);
	hipMalloc(&b, sizeof(int));
	hipMemcpy(num, a, sizeof(int)* 20, hipMemcpyHostToDevice);

	GetMaxNum_kernel << <1, 20 >> >(num, b);

	hipMemcpy(max, b, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(&num);
	hipFree(&b);

	printf("the MAX num is: %d\n", *max);

	getchar();
	return 0;
}